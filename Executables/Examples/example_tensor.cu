#include "hip/hip_runtime.h"
#include <iostream>
#include "Tensor/tensor.hpp"

#ifdef __HIPCC__

#include "Tensor/tensor.cuh"

#endif // __HIPCC__

template<typename Type>
__global__ void print_tensor(Type * data, std::size_t N){
    auto i = threadIdx.x;

    if (i != N-1 && i < N){
        printf("%f, ",data[i]);
    } else {
        printf("%f",data[i]);
    }
}

int main(){
    std::cout << "================================================================================" << std::endl
              << "Showing Tensor Interface" << std::endl
              << "================================================================================" << std::endl;

    const std::size_t N = 5;

    // cpu tensor
    NSL::Tensor<double, false> tensor_cpu(N);
#ifdef __HIPCC__
    // gpu tensor
    NSL::Tensor<double, true> tensor_gpu(N);
#endif // __HIPCC__

    std::cout << "CPU Tensor after initialization:\n T = [";
    for(std::size_t i = 0; i < N; ++i){
        if (i != N-1) {
            std::cout << tensor_cpu[i] << ", ";
        } else {
            std::cout << tensor_cpu[i];
        }
    }
    std::cout << "]" << std::endl;

#ifdef __HIPCC__

    std::cout << "GPU Tensor after initialization print from CPU:\n T = [";
    for(std::size_t i = 0; i < N; ++i){
        if (i != N-1) {
            std::cout << tensor_cpu[i] << ", ";
        } else {
            std::cout << tensor_cpu[i];
        }
    }
    std::cout << "]" << std::endl;


    std::cout << "GPU Tensor after initialization print from GPU:\n T = [";
    print_tensor<<<1,N>>>(tensor_gpu.data(),N);
    hipDeviceSynchronize();
    std::cout << "]" << std::endl;


#endif // __HIPCC__

}